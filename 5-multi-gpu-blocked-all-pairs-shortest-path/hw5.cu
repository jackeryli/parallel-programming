#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>

const int INF = ((1 << 30) - 1); 
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);
__global__ void phase1(int* Dist_gpu, int r, int n, int pitch_int);
__global__ void phase2(int* Dist_gpu, int r, int n, int pitch_int);
__global__ void phase3(int* Dist_gpu, int r, int n, int pitch_int, int thread_id, int Round);

int n, m;   // Number of vertices, edges
int* Dist;

int* Dist_gpu[2];

size_t pitch;

int N;

int main(int argc, char* argv[])
{   
    
    input(argv[1]);
    int B = 64;
    block_FW(B);
    output(argv[2]);
    hipHostFree(Dist);
    hipFree(Dist_gpu);
    return 0;
}

void input(char* infile) { 
    FILE* file = fopen(infile, "rb"); 
    fread(&n, sizeof(int), 1, file); 
    fread(&m, sizeof(int), 1, file);

    N = n;
    n = (!n%64)? n : n + 64 - n%64;

    hipHostMalloc( &Dist, sizeof(int)*(n*n), hipHostMallocDefault);

    for (int i = 0; i < n; ++i) {
        int IN = i * n;
        #pragma GCC ivdep
        for (int j = 0; j < i; ++j) {
            Dist[IN + j] = INF;
        }
        #pragma GCC ivdep
        for (int j = i + 1; j < n; ++j) {
            Dist[IN + j] = INF;
        }
    }

    int pair[3]; 
    for (int i = 0; i < m; ++i) { 
        fread(pair, sizeof(int), 3, file); 
        Dist[pair[0] * n + pair[1]] = pair[2]; 
    } 
    fclose(file); 
}

void output(char *outFileName) {
    FILE *outfile = fopen(outFileName, "w");
    for (int i = 0; i < N; ++i) {
        fwrite(&Dist[i * n], sizeof(int), N, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) {
    return (a + b -1)/b;
}

void block_FW(int B)
{
    int round = ceil(n, B);

    #pragma omp parallel num_threads(2)
    {
        int thread_id = omp_get_thread_num();
        hipSetDevice(thread_id);
        hipMalloc((void **)&Dist_gpu[thread_id], n * n * sizeof(int));
        hipMemcpy(Dist_gpu[thread_id], Dist, n * n * sizeof(int), hipMemcpyHostToDevice);
        
        dim3 grid3((round/2)+1, round);

        int pitch_int = n;

        for (int r = 0; r < round; ++r) {
            
            #pragma omp barrier
            if(r < (round/2) && thread_id == 1){
                hipMemcpyPeer((void*) &Dist_gpu[1][r * B * n], 1, (void*) &Dist_gpu[0][r * B * n], 0, B * n * sizeof(int));

            }else if(r >= (round/2) && thread_id == 0){
                if(r == (round-1))
                    hipMemcpyPeer((void*) &Dist_gpu[0][r * B * n], 0, (void*) &Dist_gpu[1][r * B * n], 1, (n - r * B) * n * sizeof(int));
                else
                    hipMemcpyPeer((void*) &Dist_gpu[0][r * B * n], 0, (void*) &Dist_gpu[1][r * B * n], 1, B * n * sizeof(int));
            }
            #pragma omp barrier

           
            phase1  <<< 1                     , dim3(32,32),   64*64*sizeof(int) >>>(Dist_gpu[thread_id], r, n, pitch_int);
            phase2  <<< dim3(round, 2)      , dim3(32,32), 2*64*64*sizeof(int) >>>(Dist_gpu[thread_id], r, n, pitch_int);
            phase3  <<< grid3, dim3(32,32), 2*64*64*sizeof(int) >>>(Dist_gpu[thread_id], r, n, pitch_int, thread_id, round);
            
                
            
            
                 
             

        }

        #pragma omp barrier
        if(thread_id == 0)
            hipMemcpy(Dist, Dist_gpu[0], (round/2) * B * n * sizeof(int), hipMemcpyDeviceToHost);
        else if(thread_id == 1)
            hipMemcpy(&Dist[(round/2) * B * n], &Dist_gpu[1][(round/2) * B * n], (n - (round/2) * B) * n * sizeof(int), hipMemcpyDeviceToHost);

    }
}

__global__ 
void phase1(int* Dist_gpu, int r, int n, int pitch_int) {

    extern __shared__ int shared_mem[]; 

    int sdx = (threadIdx.y * 64) + threadIdx.x;

    shared_mem[sdx]      = Dist_gpu[(r * 64 + threadIdx.y)     *pitch_int + r * 64 + threadIdx.x];
    shared_mem[sdx+32]   = Dist_gpu[(r * 64 + threadIdx.y)     *pitch_int + r * 64 + threadIdx.x + 32];
    shared_mem[sdx+2048] = Dist_gpu[(r * 64 + threadIdx.y + 32)*pitch_int + r * 64 + threadIdx.x];
    shared_mem[sdx+2080] = Dist_gpu[(r * 64 + threadIdx.y + 32)*pitch_int + r * 64 + threadIdx.x + 32];

    //__syncthreads();

    for(int k=0; k < 64; ++k){
        __syncthreads();
        shared_mem[sdx]      = min(shared_mem[sdx]     , shared_mem[threadIdx.y * 64 + k]    + shared_mem[k*64+threadIdx.x]);
        shared_mem[sdx+32]   = min(shared_mem[sdx+32]  , shared_mem[threadIdx.y * 64 + k]    + shared_mem[k*64+threadIdx.x + 32]);
        shared_mem[sdx+2048] = min(shared_mem[sdx+2048], shared_mem[(threadIdx.y+32)*64 + k] + shared_mem[k*64+threadIdx.x]);
        shared_mem[sdx+2080] = min(shared_mem[sdx+2080], shared_mem[(threadIdx.y+32)*64 + k] + shared_mem[k*64+threadIdx.x + 32]);
    }

    Dist_gpu[(r * 64  + threadIdx.y)     *pitch_int + r * 64 + threadIdx.x]       = shared_mem[sdx];
    Dist_gpu[(r * 64  + threadIdx.y)     *pitch_int + r * 64 + threadIdx.x + 32]  = shared_mem[sdx+32];
    Dist_gpu[(r * 64  + threadIdx.y + 32)*pitch_int + r * 64 + threadIdx.x]       = shared_mem[sdx+2048];
    Dist_gpu[(r * 64  + threadIdx.y + 32)*pitch_int + r * 64 + threadIdx.x + 32]  = shared_mem[sdx+2080];
}

__global__ void phase2(int* Dist_gpu, int r, int n, int pitch_int) {

    extern __shared__ int shared_mem[];

    int sdx = threadIdx.y * 64 + threadIdx.x;
    int b_i, b_j, i, j;

    if(blockIdx.y == 0){
        b_i = blockIdx.x;
        b_j = r;

        if(b_i == r) return;

        i = b_i * 64 + threadIdx.y;
        j = b_j * 64 + threadIdx.x;

        shared_mem[sdx]                    = Dist_gpu[i                             * pitch_int + j      ]; // IK
        shared_mem[sdx + 32]               = Dist_gpu[i                             * pitch_int + j + 32 ]; // IK
        shared_mem[sdx + 4096]             = Dist_gpu[(r*64 + threadIdx.y)      * pitch_int + j      ]; // KJ
        shared_mem[sdx + 4128]             = Dist_gpu[(r*64 + threadIdx.y)      * pitch_int + j + 32 ]; // KJ

        shared_mem[sdx + 2048]             = Dist_gpu[(i + 32)                      * pitch_int + j      ];
        shared_mem[sdx + 2080]             = Dist_gpu[(i + 32)                      * pitch_int + j + 32 ];
        shared_mem[sdx + 2048 + 4096]      = Dist_gpu[(r*64 + threadIdx.y + 32) * pitch_int + j      ];
        shared_mem[sdx + 2080 + 4096]      = Dist_gpu[(r*64 + threadIdx.y + 32) * pitch_int + j + 32 ];

        #pragma unroll
        for (int k = 0; k < 64; ++k) {
            __syncthreads();
            
            shared_mem[sdx]      = min(shared_mem[sdx],      shared_mem[threadIdx.y*64+k] + shared_mem[k*64+threadIdx.x + 4096]);
            shared_mem[sdx + 32] = min(shared_mem[sdx + 32], shared_mem[threadIdx.y*64+k] + shared_mem[k*64+threadIdx.x + 4128]);

            shared_mem[sdx + 2048] = min(shared_mem[sdx+2048], shared_mem[(threadIdx.y + 32)*64+k] + shared_mem[k*64+threadIdx.x + 4096]);
            shared_mem[sdx + 2080] = min(shared_mem[sdx+2080], shared_mem[(threadIdx.y + 32)*64+k] + shared_mem[k*64+threadIdx.x + 4128]);
        }

        Dist_gpu[i       *pitch_int + j     ] = shared_mem[sdx            ];  
        Dist_gpu[i       *pitch_int + j + 32] = shared_mem[sdx + 32       ];
        Dist_gpu[(i + 32)*pitch_int + j     ] = shared_mem[sdx + 2048     ];  
        Dist_gpu[(i + 32)*pitch_int + j + 32] = shared_mem[sdx + 2048 + 32];
    }else if(blockIdx.y == 1){
        b_i = r;
        b_j = blockIdx.x;

        if(b_j == r) return;

        i = b_i * 64 + threadIdx.y;
        j = b_j * 64 + threadIdx.x;

        shared_mem[sdx]                    = Dist_gpu[i*pitch_int + j];
        shared_mem[sdx + 32]               = Dist_gpu[i*pitch_int + j + 32];
        shared_mem[sdx + 4096]             = Dist_gpu[i*pitch_int + r * 64 + threadIdx.x];
        shared_mem[sdx + 4096 + 32]        = Dist_gpu[i*pitch_int + r * 64 + threadIdx.x + 32];

        shared_mem[sdx + 2048]             = Dist_gpu[(i + 32)*pitch_int + j];
        shared_mem[sdx + 2080]             = Dist_gpu[(i + 32)*pitch_int + j + 32];
        shared_mem[sdx + 2048 + 4096]      = Dist_gpu[(i + 32)*pitch_int + r * 64 + threadIdx.x];
        shared_mem[sdx + 2080 + 4096]      = Dist_gpu[(i + 32)*pitch_int + r * 64 + threadIdx.x + 32];



        #pragma unroll
        for (int k = 0; k < 64; ++k) {
            __syncthreads();

            
            shared_mem[sdx] = min(shared_mem[sdx], shared_mem[threadIdx.y*64+k+4096] + shared_mem[k*64+threadIdx.x]);
            shared_mem[sdx + 32] = min(shared_mem[sdx + 32], shared_mem[threadIdx.y*64+k+4096] + shared_mem[k*64+threadIdx.x + 32]);

            
            shared_mem[sdx+2048] = min(shared_mem[sdx+2048], shared_mem[(threadIdx.y+32)*64+k+4096] + shared_mem[k*64+threadIdx.x]);
            shared_mem[sdx+2080] = min(shared_mem[sdx+2080], shared_mem[(threadIdx.y+32)*64+k+4096] + shared_mem[k*64+threadIdx.x + 32]);
            
        }

        Dist_gpu[i*pitch_int + j]      = shared_mem[sdx];  
        Dist_gpu[i*pitch_int + j + 32] = shared_mem[sdx + 32];

        Dist_gpu[(i + 32)*pitch_int + j]      = shared_mem[sdx+2048];  
        Dist_gpu[(i + 32)*pitch_int + j + 32] = shared_mem[sdx+2048 + 32];
    }
    
    

    
}

__global__ void phase3(int* Dist_gpu, int r, int n, int pitch_int, int thread_id, int Round) {

    int b_i = blockIdx.x;
    int b_j = blockIdx.y;

    if(thread_id == 1) b_i += (Round/2);
   
    if(b_i == r || b_j == r) return;
    if(b_i == r || b_j == r) return;
    
    int i = b_i * 64 + threadIdx.y;
    int j = b_j * 64 + threadIdx.x;

    if (i >= n || j >= n) return;

    extern __shared__ int shared_mem[];
    
    int d0 = Dist_gpu[i*pitch_int + j];
    int d1 = Dist_gpu[i*pitch_int + j + 32];
    int d2 = Dist_gpu[(i+32)*pitch_int + j];
    int d3 = Dist_gpu[(i+32)*pitch_int + j + 32];
    
    int sdx = threadIdx.y * 64 + threadIdx.x;

    shared_mem[ sdx ]       = Dist_gpu[i*pitch_int + r * 64 + threadIdx.x];
    shared_mem[ sdx + 32]   = Dist_gpu[i*pitch_int + r * 64 + threadIdx.x + 32];
    shared_mem[ sdx + 4096] = Dist_gpu[(r * 64 + threadIdx.y)*pitch_int + j];
    shared_mem[ sdx + 4128] = Dist_gpu[(r * 64 + threadIdx.y)*pitch_int + j + 32];

    sdx += 2048;

    shared_mem[ sdx ]       = Dist_gpu[(i + 32)*pitch_int + r * 64 + threadIdx.x];
    shared_mem[ sdx + 32]   = Dist_gpu[(i + 32)*pitch_int + r * 64 + threadIdx.x + 32];
    shared_mem[ sdx + 4096] = Dist_gpu[(r * 64 + threadIdx.y + 32)*pitch_int + j];
    shared_mem[ sdx + 4128] = Dist_gpu[(r * 64 + threadIdx.y + 32)*pitch_int + j + 32];

    __syncthreads();
    
    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        int idx = threadIdx.y * 64 + k;
        int v0 = shared_mem[idx]        + shared_mem[k*64 + threadIdx.x + 4096];
        int v1 = shared_mem[idx]        + shared_mem[k*64 + threadIdx.x + 4128];
        int v2 = shared_mem[idx + 2048] + shared_mem[k*64 + threadIdx.x + 4096];
        int v3 = shared_mem[idx + 2048] + shared_mem[k*64 + threadIdx.x + 4128];
        d0 = min(d0, v0);
        d1 = min(d1, v1);
        d2 = min(d2, v2);
        d3 = min(d3, v3);
    }

    Dist_gpu[i*pitch_int + j]           = d0;
    Dist_gpu[i*pitch_int + j + 32]      = d1;
    Dist_gpu[(i+32)*pitch_int + j]      = d2;
    Dist_gpu[(i+32)*pitch_int + j + 32] = d3;

}
